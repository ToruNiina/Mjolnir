#include "hip/hip_runtime.h"
#include <mjolnir/cuda/BAOABLangevinIntegrator.hpp>
#include <mjolnir/cuda/Vector.hpp>
#include <thrust/tuple.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

namespace mjolnir
{

// ---------------------------------------------------------------------------
// To reduce implementation cost, first implement template function.
// This cannot be written in a header file because it uses cuda device
// operations and the header file would be included from .cpp file that will be
// compiled by gcc. When this template function is instanciated via cpp file,
// gcc cannot find the way to compile cuda operations and stop with an error.
// Therefore, we need to write all the definitions in a .cu file.

namespace detail
{

template<typename realT, template<typename, typename> class boundaryT>
realT cuda_BAOABLangevinIntegrator_step1(
    BAOABLangevinIntegrator<CUDASimulatorTraits<realT, boundaryT>>& integ,
    System                 <CUDASimulatorTraits<realT, boundaryT>>& sys,
    RandomNumberGenerator  <CUDASimulatorTraits<realT, boundaryT>>& rng) noexcept
{
    using namespace ::mjolnir::math; // for real4 operators
    using traits_type     = CUDASimulatorTraits<realT, UnlimitedBoundary>;
    using real_type       = typename traits_type::real_type;
    using coordinate_type = typename traits_type::coordinate_type;

    rng.fill_with_gaussian(integ.random_forces_device());
    thrust::device_ptr<const real_type> rnd = integ.random_forces_device().data();

    const real_type kB     = physics::constants<real_type>::kB();
    const real_type T_ref  = sys.attribute("temperature");
    const real_type kBT    = kB * T_ref;
    const real_type halfdt = integ.halfdt();

    const thrust::device_ptr<const real_type> expgts  = integ.exp_gamma_dt_device().data();
    const thrust::device_ptr<const real_type> noises  = integ.noise_coeff_device().data();
    const thrust::device_ptr<const real_type> rmasses = sys.rmasses_device().data();
    const thrust::device_ptr<coordinate_type> ps      = sys.positions_device().data();
    const thrust::device_ptr<coordinate_type> vs      = sys.velocities_device().data();
    const thrust::device_ptr<coordinate_type> fs      = sys.forces_device().data();

    const auto boundary = sys.boundary();

    return thrust::transform_reduce(thrust::device,
        thrust::counting_iterator<std::size_t>(0),
        thrust::counting_iterator<std::size_t>(sys.size()),
        [halfdt, kBT, rnd, expgts, noises, rmasses, ps, vs, fs, boundary]
        __device__ (const std::size_t idx)
        {
            const std::size_t offset = idx * 3;
            coordinate_type R;
            R.x = rnd.get()[offset+0];
            R.y = rnd.get()[offset+1];
            R.z = rnd.get()[offset+2];
            R.w = 0;

            const real_type  expgt = expgts .get()[idx];
            const real_type  noise = noises .get()[idx];
            const real_type  rm    = rmasses.get()[idx];
            coordinate_type& p     = ps.get()[idx];
            coordinate_type& v     = vs.get()[idx];
            coordinate_type& f     = fs.get()[idx];
            coordinate_type  dp{0,0,0,0};

            v  += halfdt * rm * f; // calc v(n+1/3)
            dp += halfdt * v;      // calc p(n+1/2)
            v  *= expgt;
            v  += noise  * R;      // calc v(n+2/3)
            dp += halfdt * v;      // calc p(n+1)

            // update p(n) -> p(n+1);
            p  = boundary.adjust_position(p + dp);

            // reset force
            f = coordinate_type{0,0,0,0};

            // collect largest displacement
            return dp.x * dp.x + dp.y * dp.y + dp.z * dp.z;

        }, 0.0, thrust::maximum<double>());
}

template<typename traitsT>
void cuda_BAOABLangevinIntegrator_step2(BAOABLangevinIntegrator<traitsT>& integ,
                                        System<traitsT>& sys) noexcept
{
    using traits_type     = traitsT;
    using real_type       = typename traits_type::real_type;
    using coordinate_type = typename traits_type::coordinate_type;

    const real_type halfdt = integ.halfdt();
    const thrust::device_ptr<const real_type> rmasses = sys.rmasses_device().data();
    const thrust::device_ptr<coordinate_type> vs      = sys.velocities_device().data();
    const thrust::device_ptr<coordinate_type> fs      = sys.forces_device().data();

    // calc v(n+2/3) -> v(n+1)
    thrust::for_each(thrust::device,
        thrust::counting_iterator<std::size_t>(0),
        thrust::counting_iterator<std::size_t>(sys.size()),
        [halfdt, rmasses, vs, fs] __device__ (const std::size_t idx)
        {
            vs.get()[idx] += halfdt * rmasses[idx] * fs[idx];
            return;
        });
    return ;
}

template<typename traitsT>
void cuda_BAOABLangevinIntegrator_update_parameters(
        BAOABLangevinIntegrator<traitsT>& integ, const System<traitsT>& sys) noexcept
{
    using traits_type     = traitsT;
    using real_type       = typename traits_type::real_type;
    using coordinate_type = typename traits_type::coordinate_type;

    const real_type dt  = integ.delta_t();
    const real_type kBT = physics::constants<real_type>::kB() * integ.temperature();

    const thrust::device_ptr<const real_type> rmasses = sys.rmasses_device().data();
    const thrust::device_ptr<const real_type> gammas  = integ.gammas_device().data();
    const thrust::device_ptr<real_type>       expgts  = integ.exp_gamma_dt_device().data();
    const thrust::device_ptr<real_type>       noises  = integ.noise_coeff_device().data();

    thrust::for_each(thrust::device,
        thrust::counting_iterator<std::size_t>(0),
        thrust::counting_iterator<std::size_t>(sys.size()),
        [dt, kBT, rmasses, gammas, expgts, noises]
        __device__ (const std::size_t idx)
        {
            const real_type rmass    = rmasses[idx];
            const real_type gamma_dt = -real_type(1) * gammas[idx] * dt;
            expgts[idx] = ::exp(gamma_dt);
            noises[idx] = ::sqrt(kBT * (real_type(1) - ::exp(real_type(2) * gamma_dt)) * rmass);
            return ;
        });
    return ;
}
} // detail

// ----------------------------------------------------------------------------
// definitions of BAOABLangevinIntegrator::step()

template<>
typename BAOABLangevinIntegrator<CUDASimulatorTraits<double, UnlimitedBoundary>>::real_type
BAOABLangevinIntegrator<CUDASimulatorTraits<double, UnlimitedBoundary>>::step(
    const real_type time, system_type& sys, forcefield_type& ff, rng_type& rng)
{
    const real_type largest_disp2 =
        detail::cuda_BAOABLangevinIntegrator_step1(*this, sys, rng);

    // update neighbor list; reduce margin, reconstruct the list if needed
    ff.reduce_margin(2 * std::sqrt(largest_disp2), sys);

    // calc f(p(n+1))
    ff.calc_force(sys);

    // calc v(n+2/3) -> v(n+1)
    detail::cuda_BAOABLangevinIntegrator_step2(*this, sys);

    // XXX Note that the updated configurations are not pulled here.
    //     Almost all the computations are done on GPU in this implemnetation.
    //     Observers require the values, so they pull it back to CPU.

    return time + dt_;
}

template<>
typename BAOABLangevinIntegrator<CUDASimulatorTraits<float, UnlimitedBoundary>>::real_type
BAOABLangevinIntegrator<CUDASimulatorTraits<float, UnlimitedBoundary>>::step(
    const real_type time, system_type& sys, forcefield_type& ff, rng_type& rng)
{
    const real_type largest_disp2 =
        detail::cuda_BAOABLangevinIntegrator_step1(*this, sys, rng);

    // update neighbor list; reduce margin, reconstruct the list if needed
    ff.reduce_margin(2 * std::sqrt(largest_disp2), sys);

    // calc f(p(n+1))
    ff.calc_force(sys);

    // calc v(n+2/3) -> v(n+1)
    detail::cuda_BAOABLangevinIntegrator_step2(*this, sys);

    // XXX Note that the updated configurations are not pulled here.
    //     Almost all the computations are done on GPU in this implemnetation.
    //     Observers require the values, so they pull it back to CPU.

    return time + dt_;
}

template<>
typename BAOABLangevinIntegrator<CUDASimulatorTraits<double, CuboidalPeriodicBoundary>>::real_type
BAOABLangevinIntegrator<CUDASimulatorTraits<double, CuboidalPeriodicBoundary>>::step(
    const real_type time, system_type& sys, forcefield_type& ff, rng_type& rng)
{
    const real_type largest_disp2 =
        detail::cuda_BAOABLangevinIntegrator_step1(*this, sys, rng);

    // update neighbor list; reduce margin, reconstruct the list if needed
    ff.reduce_margin(2 * std::sqrt(largest_disp2), sys);

    // calc f(p(n+1))
    ff.calc_force(sys);

    // calc v(n+2/3) -> v(n+1)
    detail::cuda_BAOABLangevinIntegrator_step2(*this, sys);

    // XXX Note that the updated configurations are not pulled here.
    //     Almost all the computations are done on GPU in this implemnetation.
    //     Observers require the values, so they pull it back to CPU.

    return time + dt_;
}

template<>
typename BAOABLangevinIntegrator<CUDASimulatorTraits<float, CuboidalPeriodicBoundary>>::real_type
BAOABLangevinIntegrator<CUDASimulatorTraits<float, CuboidalPeriodicBoundary>>::step(
    const real_type time, system_type& sys, forcefield_type& ff, rng_type& rng)
{
    const real_type largest_disp2 =
        detail::cuda_BAOABLangevinIntegrator_step1(*this, sys, rng);

    // update neighbor list; reduce margin, reconstruct the list if needed
    ff.reduce_margin(2 * std::sqrt(largest_disp2), sys);

    // calc f(p(n+1))
    ff.calc_force(sys);

    // calc v(n+2/3) -> v(n+1)
    detail::cuda_BAOABLangevinIntegrator_step2(*this, sys);

    // XXX Note that the updated configurations are not pulled here.
    //     Almost all the computations are done on GPU in this implemnetation.
    //     Observers require the values, so they pull it back to CPU.

    return time + dt_;
}

// ---------------------------------------------------------------------------
// definitions of BAOABLangevinIntegrator<CUDA>::initialize();

template<>
void BAOABLangevinIntegrator<CUDASimulatorTraits<double, UnlimitedBoundary>>::initialize(
        system_type& sys, forcefield_type& ff, rng_type&)
{
    // calculate parameters for each particles
    this->update(sys);

    // zero-clear force
    for(std::size_t i=0; i<sys.size(); ++i)
    {
        sys.force(i) = math::make_coordinate<coordinate_type>(0, 0, 0);
    }
    // also on the device.
    sys.forces_device() = sys.forces_host();

    // calculate the current force.
    ff.calc_force(sys);
    return;
}
template<>
void BAOABLangevinIntegrator<CUDASimulatorTraits<float, UnlimitedBoundary>>::initialize(
        system_type& sys, forcefield_type& ff, rng_type&)
{
    // calculate parameters for each particles
    this->update(sys);

    // zero-clear force
    for(std::size_t i=0; i<sys.size(); ++i)
    {
        sys.force(i) = math::make_coordinate<coordinate_type>(0, 0, 0);
    }
    // also on the device.
    sys.forces_device() = sys.forces_host();

    // calculate the current force.
    ff.calc_force(sys);
    return;
}
template<>
void BAOABLangevinIntegrator<CUDASimulatorTraits<double, CuboidalPeriodicBoundary>>::initialize(
        system_type& sys, forcefield_type& ff, rng_type&)
{
    // calculate parameters for each particles
    this->update(sys);

    // zero-clear force
    for(std::size_t i=0; i<sys.size(); ++i)
    {
        sys.force(i) = math::make_coordinate<coordinate_type>(0, 0, 0);
    }
    // also on the device.
    sys.forces_device() = sys.forces_host();

    // calculate the current force.
    ff.calc_force(sys);
    return;
}
template<>
void BAOABLangevinIntegrator<CUDASimulatorTraits<float, CuboidalPeriodicBoundary>>::initialize(
        system_type& sys, forcefield_type& ff, rng_type&)
{
    // calculate parameters for each particles
    this->update(sys);

    // zero-clear force
    for(std::size_t i=0; i<sys.size(); ++i)
    {
        sys.force(i) = math::make_coordinate<coordinate_type>(0, 0, 0);
    }
    // also on the device.
    sys.forces_device() = sys.forces_host();

    // calculate the current force.
    ff.calc_force(sys);
    return;
}

// ---------------------------------------------------------------------------
// definitions of BAOABLangevinIntegrator<CUDA>::reset_parameters();

template<>
void BAOABLangevinIntegrator<CUDASimulatorTraits<double, UnlimitedBoundary>
    >::reset_parameters(const system_type& sys)
{
    detail::cuda_BAOABLangevinIntegrator_update_parameters(*this, sys);
    return;
}

template<>
void BAOABLangevinIntegrator<CUDASimulatorTraits<float, UnlimitedBoundary>
    >::reset_parameters(const system_type& sys)
{
    detail::cuda_BAOABLangevinIntegrator_update_parameters(*this, sys);
    return;
}

template<>
void BAOABLangevinIntegrator<CUDASimulatorTraits<double, CuboidalPeriodicBoundary>
    >::reset_parameters(const system_type& sys)
{
    detail::cuda_BAOABLangevinIntegrator_update_parameters(*this, sys);
    return;
}

template<>
void BAOABLangevinIntegrator<CUDASimulatorTraits<float, CuboidalPeriodicBoundary>
    >::reset_parameters(const system_type& sys)
{
    detail::cuda_BAOABLangevinIntegrator_update_parameters(*this, sys);
    return;
}

// ----------------------------------------------------------------------------
// any constructors and `operator=`s are the same as the one that will be
// generated by `= default`, but it should be here because those uses
// `thrust::device_vector`s copy/move constructors and `operator=`s that uses
// device instructions.
//     Since all the implementations are the same, it uses macro.

#define GENERATE_BAOAB_LANGEVIN_INTEGRATOR_CONSTRUCTOR_INSTANCES(REAL, BOUNDARY)          \
    template<>                                                                            \
    BAOABLangevinIntegrator<CUDASimulatorTraits<REAL, BOUNDARY>>::BAOABLangevinIntegrator(\
            const real_type dt, const std::vector<real_type>& gamma)                      \
        : dt_(dt), halfdt_(dt / 2), gammas_h_(gamma.begin(), gamma.end()),                \
          gammas_(gammas_h_), exp_gamma_dt_(gammas_.size()),                              \
          noise_coeff_ (gammas_.size()), random_forces_(gammas_.size() * 3)               \
    {} /**/

GENERATE_BAOAB_LANGEVIN_INTEGRATOR_CONSTRUCTOR_INSTANCES(double, UnlimitedBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_CONSTRUCTOR_INSTANCES(float,  UnlimitedBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_CONSTRUCTOR_INSTANCES(double, CuboidalPeriodicBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_CONSTRUCTOR_INSTANCES(float,  CuboidalPeriodicBoundary)

#undef GENERATE_BAOAB_LANGEVIN_INTEGRATOR_CONSTRUCTOR_INSTANCES

#define GENERATE_BAOAB_LANGEVIN_INTEGRATOR_DESTRUCTOR_INSTANCES(REAL, BOUNDARY)             \
    template<>                                                                              \
    BAOABLangevinIntegrator<CUDASimulatorTraits<REAL, BOUNDARY>>::~BAOABLangevinIntegrator()\
    {} /**/

GENERATE_BAOAB_LANGEVIN_INTEGRATOR_DESTRUCTOR_INSTANCES(double, UnlimitedBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_DESTRUCTOR_INSTANCES(float,  UnlimitedBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_DESTRUCTOR_INSTANCES(double, CuboidalPeriodicBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_DESTRUCTOR_INSTANCES(float,  CuboidalPeriodicBoundary)

#undef GENERATE_BAOAB_LANGEVIN_INTEGRATOR_DESTRUCTOR_INSTANCES

#define GENERATE_BAOAB_LANGEVIN_INTEGRATOR_COPY_CONSTRUCTOR_INSTANCES(REAL, BOUNDARY)     \
    template<>                                                                            \
    BAOABLangevinIntegrator<CUDASimulatorTraits<REAL, BOUNDARY>>::BAOABLangevinIntegrator(\
            const BAOABLangevinIntegrator& other)                                         \
        : dt_(other.dt_), halfdt_(other.halfdt_), gammas_h_(other.gammas_h_),             \
          gammas_(other.gammas_), exp_gamma_dt_(other.exp_gamma_dt_),                     \
          noise_coeff_ (other.noise_coeff_), random_forces_(other.random_forces_)         \
    {} /**/

GENERATE_BAOAB_LANGEVIN_INTEGRATOR_COPY_CONSTRUCTOR_INSTANCES(double, UnlimitedBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_COPY_CONSTRUCTOR_INSTANCES(float,  UnlimitedBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_COPY_CONSTRUCTOR_INSTANCES(double, CuboidalPeriodicBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_COPY_CONSTRUCTOR_INSTANCES(float,  CuboidalPeriodicBoundary)

#undef GENERATE_BAOAB_LANGEVIN_INTEGRATOR_COPY_CONSTRUCTOR_INSTANCES

#define GENERATE_BAOAB_LANGEVIN_INTEGRATOR_MOVE_CONSTRUCTOR_INSTANCES(REAL, BOUNDARY)      \
    template<>                                                                             \
    BAOABLangevinIntegrator<CUDASimulatorTraits<REAL, BOUNDARY>>::BAOABLangevinIntegrator( \
            BAOABLangevinIntegrator&& other)                                               \
        : dt_(other.dt_), halfdt_(other.halfdt_), gammas_h_(std::move(other.gammas_h_)),   \
          gammas_(std::move(other.gammas_)), exp_gamma_dt_(std::move(other.exp_gamma_dt_)),\
          noise_coeff_ (std::move(other.noise_coeff_)),                                    \
          random_forces_(std::move(other.random_forces_))                                  \
    {} /**/

GENERATE_BAOAB_LANGEVIN_INTEGRATOR_MOVE_CONSTRUCTOR_INSTANCES(double, UnlimitedBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_MOVE_CONSTRUCTOR_INSTANCES(float,  UnlimitedBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_MOVE_CONSTRUCTOR_INSTANCES(double, CuboidalPeriodicBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_MOVE_CONSTRUCTOR_INSTANCES(float,  CuboidalPeriodicBoundary)

#undef GENERATE_BAOAB_LANGEVIN_INTEGRATOR_MOVE_CONSTRUCTOR_INSTANCES

#define GENERATE_BAOAB_LANGEVIN_INTEGRATOR_COPY_ASSIGNMENT_INSTANCES(REAL, BOUNDARY)\
    template<>                                                                      \
    BAOABLangevinIntegrator<CUDASimulatorTraits<REAL, BOUNDARY>>&                   \
    BAOABLangevinIntegrator<CUDASimulatorTraits<REAL, BOUNDARY>>::operator=(        \
            const BAOABLangevinIntegrator& other)                                   \
    {                                                                               \
        this->dt_            = other.dt_;                                           \
        this->halfdt_        = other.halfdt_;                                       \
        this->gammas_h_      = other.gammas_h_;                                     \
        this->gammas_        = other.gammas_;                                       \
        this->exp_gamma_dt_  = other.exp_gamma_dt_;                                 \
        this->noise_coeff_   = other.noise_coeff_;                                  \
        this->random_forces_ = other.random_forces_;                                \
        return *this;                                                               \
    } /**/

GENERATE_BAOAB_LANGEVIN_INTEGRATOR_COPY_ASSIGNMENT_INSTANCES(double, UnlimitedBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_COPY_ASSIGNMENT_INSTANCES(float,  UnlimitedBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_COPY_ASSIGNMENT_INSTANCES(double, CuboidalPeriodicBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_COPY_ASSIGNMENT_INSTANCES(float,  CuboidalPeriodicBoundary)

#undef GENERATE_BAOAB_LANGEVIN_INTEGRATOR_COPY_ASSIGNMENT_INSTANCES

#define GENERATE_BAOAB_LANGEVIN_INTEGRATOR_MOVE_ASSIGNMENT_INSTANCES(REAL, BOUNDARY)\
    template<>                                                                      \
    BAOABLangevinIntegrator<CUDASimulatorTraits<REAL, BOUNDARY>>&                   \
    BAOABLangevinIntegrator<CUDASimulatorTraits<REAL, BOUNDARY>>::operator=(        \
            BAOABLangevinIntegrator&& other)                                        \
    {                                                                               \
        this->dt_            = std::move(other.dt_);                                \
        this->halfdt_        = std::move(other.halfdt_);                            \
        this->gammas_h_      = std::move(other.gammas_h_);                          \
        this->gammas_        = std::move(other.gammas_);                            \
        this->exp_gamma_dt_  = std::move(other.exp_gamma_dt_);                      \
        this->noise_coeff_   = std::move(other.noise_coeff_);                       \
        this->random_forces_ = std::move(other.random_forces_);                     \
        return *this;                                                               \
    } /**/

GENERATE_BAOAB_LANGEVIN_INTEGRATOR_MOVE_ASSIGNMENT_INSTANCES(double, UnlimitedBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_MOVE_ASSIGNMENT_INSTANCES(float,  UnlimitedBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_MOVE_ASSIGNMENT_INSTANCES(double, CuboidalPeriodicBoundary)
GENERATE_BAOAB_LANGEVIN_INTEGRATOR_MOVE_ASSIGNMENT_INSTANCES(float,  CuboidalPeriodicBoundary)

#undef GENERATE_BAOAB_LANGEVIN_INTEGRATOR_MOVE_ASSIGNMENT_INSTANCES

// ---------------------------------------------------------------------------
// class template instanciation

template class BAOABLangevinIntegrator<CUDASimulatorTraits<double, UnlimitedBoundary>>;
template class BAOABLangevinIntegrator<CUDASimulatorTraits<float,  UnlimitedBoundary>>;
template class BAOABLangevinIntegrator<CUDASimulatorTraits<double, CuboidalPeriodicBoundary>>;
template class BAOABLangevinIntegrator<CUDASimulatorTraits<float,  CuboidalPeriodicBoundary>>;

} // mjolnir

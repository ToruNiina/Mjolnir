#include <mjolnir/cuda/RandomNumberGenerator.hpp>

namespace mjolnir
{
namespace detail
{
hiprandStatus_t curand_generate_uniform_01(hiprandGenerator_t gen, thrust::device_vector<double>& buf)
{
    return hiprandGenerateUniformDouble(gen, buf.data().get(), buf.size());
}
hiprandStatus_t curand_generate_uniform_01(hiprandGenerator_t gen, thrust::device_vector<float>& buf)
{
    return hiprandGenerateUniform(gen, buf.data().get(), buf.size());
}
hiprandStatus_t curand_generate_gaussian(hiprandGenerator_t gen, thrust::device_vector<double>& buf)
{
    return hiprandGenerateNormalDouble(
            gen, buf.data().get(), buf.size(), /*mean*/ 0.0, /*stddev*/ 1.0);
}
hiprandStatus_t curand_generate_gaussian(hiprandGenerator_t gen, thrust::device_vector<float>& buf)
{
    return hiprandGenerateNormal(
            gen, buf.data().get(), buf.size(), /*mean*/ 0.0, /*stddev*/ 1.0);
}
} // detail

// ---------------------------------------------------------------------------
// class template instanciation
//

template class RandomNumberGenerator<CUDASimulatorTraits<double, UnlimitedBoundary>>;
template class RandomNumberGenerator<CUDASimulatorTraits<float,  UnlimitedBoundary>>;
template class RandomNumberGenerator<CUDASimulatorTraits<double, CuboidalPeriodicBoundary>>;
template class RandomNumberGenerator<CUDASimulatorTraits<float,  CuboidalPeriodicBoundary>>;
}

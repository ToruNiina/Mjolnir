#include "hip/hip_runtime.h"
#include <mjolnir/cuda/System.hpp>
#include <mjolnir/cuda/Vector.hpp>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_ptr.h>
#include <hip/hip_runtime.h>

namespace mjolnir
{

// ---------------------------------------------------------------------------
// definitions of BAOABLangevinIntegrator<CUDA>::step();

namespace detail
{
template<typename traitsT>
void cuda_system_initialize(
        System<traitsT>& sys, RandomNumberGenerator<traitsT>& rng)
{
    MJOLNIR_GET_DEFAULT_LOGGER();
    MJOLNIR_LOG_FUNCTION();

    using real_type       = typename traitsT::real_type;
    using coordinate_type = typename traitsT::coordinate_type;

    const real_type kB    = physics::constants<real_type>::kB();
    const real_type T_ref = sys.attribute("temperature");
    const real_type kBT   = kB * T_ref;

    MJOLNIR_LOG_NOTICE("generating velocity with T = ", T_ref, "...");

    // generate gaussians using cuRAND

    thrust::device_vector<real_type> gaussians(sys.size() * 3);
    rng.fill_with_gaussian(gaussians);

    thrust::device_ptr<const real_type> rmass = sys.rmasses_device().data();
    thrust::device_ptr<coordinate_type> velo  = sys.velocities_device().data();
    thrust::device_ptr<const real_type> rnd   = gaussians.data();

    // generate Maxwell-Boltzmann distribution
    thrust::for_each(thrust::device,
        thrust::counting_iterator<std::size_t>(0),
        thrust::counting_iterator<std::size_t>(sys.size()),
        [kBT, rmass, velo, rnd] __device__ (const std::size_t idx)
        {
            const std::size_t offset = idx * 3;
            const auto R = math::make_coordinate<coordinate_type>(
                    rnd[offset], rnd[offset+1], rnd[offset+2]);

            velo[idx] = ::sqrt(kBT * rmass[idx]) * R;
            return;
        });

    // pull back generated velocities to system
    sys.velocities_host() = sys.velocities_device();

    MJOLNIR_LOG_NOTICE("done.");
    return;
}

} // detail

template<>
void System<CUDASimulatorTraits<double, UnlimitedBoundary>>::initialize(rng_type& rng)
{
    MJOLNIR_GET_DEFAULT_LOGGER();
    MJOLNIR_LOG_FUNCTION();

    // copy initial conditions from CPU to GPU
    this->masses_d_     = this->masses_    ;
    this->rmasses_d_    = this->rmasses_   ;
    this->positions_d_  = this->positions_ ;
    this->velocities_d_ = this->velocities_;
    this->forces_d_     = this->forces_    ;

    if(this->velocity_initialized_)
    {
        MJOLNIR_LOG_NOTICE(
            "velocity is already given, nothing to initialize in System");
        return ;
    }
    assert(this->has_attribute("temperature"));

    detail::cuda_system_initialize(*this, rng);
    return;
}

template<>
void System<CUDASimulatorTraits<float, UnlimitedBoundary>>::initialize(rng_type& rng)
{
    MJOLNIR_GET_DEFAULT_LOGGER();
    MJOLNIR_LOG_FUNCTION();

    // copy initial conditions from CPU to GPU
    this->masses_d_     = this->masses_    ;
    this->rmasses_d_    = this->rmasses_   ;
    this->positions_d_  = this->positions_ ;
    this->velocities_d_ = this->velocities_;
    this->forces_d_     = this->forces_    ;

    if(this->velocity_initialized_)
    {
        MJOLNIR_LOG_NOTICE(
            "velocity is already given, nothing to initialize in System");
        return ;
    }
    assert(this->has_attribute("temperature"));

    detail::cuda_system_initialize(*this, rng);
    return;
}

template<>
void System<CUDASimulatorTraits<double, CuboidalPeriodicBoundary>>::initialize(rng_type& rng)
{
    MJOLNIR_GET_DEFAULT_LOGGER();
    MJOLNIR_LOG_FUNCTION();

    // copy initial conditions from CPU to GPU
    this->masses_d_     = this->masses_    ;
    this->rmasses_d_    = this->rmasses_   ;
    this->positions_d_  = this->positions_ ;
    this->velocities_d_ = this->velocities_;
    this->forces_d_     = this->forces_    ;

    if(this->velocity_initialized_)
    {
        MJOLNIR_LOG_NOTICE(
            "velocity is already given, nothing to initialize in System");
        return ;
    }
    assert(this->has_attribute("temperature"));

    detail::cuda_system_initialize(*this, rng);
    return;
}

template<>
void System<CUDASimulatorTraits<float, CuboidalPeriodicBoundary>>::initialize(rng_type& rng)
{
    MJOLNIR_GET_DEFAULT_LOGGER();
    MJOLNIR_LOG_FUNCTION();

    // copy initial conditions from CPU to GPU
    this->masses_d_     = this->masses_    ;
    this->rmasses_d_    = this->rmasses_   ;
    this->positions_d_  = this->positions_ ;
    this->velocities_d_ = this->velocities_;
    this->forces_d_     = this->forces_    ;

    if(this->velocity_initialized_)
    {
        MJOLNIR_LOG_NOTICE(
            "velocity is already given, nothing to initialize in System");
        return ;
    }
    assert(this->has_attribute("temperature"));

    detail::cuda_system_initialize(*this, rng);
    return;
}

template<>
void System<CUDASimulatorTraits<double, UnlimitedBoundary>>::sync_configurations()
{
    // assuming that masses will not be changed ...
    this->positions_  = this->positions_d_ ;
    this->velocities_ = this->velocities_d_;
    this->forces_     = this->forces_d_    ;
    return;
}

template<>
void System<CUDASimulatorTraits<float, UnlimitedBoundary>>::sync_configurations()
{
    // assuming that masses will not be changed ...
    this->positions_  = this->positions_d_ ;
    this->velocities_ = this->velocities_d_;
    this->forces_     = this->forces_d_    ;
    return;
}

template<>
void System<CUDASimulatorTraits<double, CuboidalPeriodicBoundary>>::sync_configurations()
{
    // assuming that masses will not be changed ...
    this->positions_  = this->positions_d_ ;
    this->velocities_ = this->velocities_d_;
    this->forces_     = this->forces_d_    ;
    return;
}

template<>
void System<CUDASimulatorTraits<float, CuboidalPeriodicBoundary>>::sync_configurations()
{
    // assuming that masses will not be changed ...
    this->positions_  = this->positions_d_ ;
    this->velocities_ = this->velocities_d_;
    this->forces_     = this->forces_d_    ;
    return;
}

// ----------------------------------------------------------------------------
// any constructors and `operator=`s are the same as the one that will be
// generated by `= default`, but it should be here because those uses
// `thrust::device_vector`s copy/move constructors and `operator=`s that uses
// device instructions.
//     Since all the implementations are the same, it uses macro.

#define GENERATE_SYSTEM_CONSTRUCTOR_INSTANCES(REAL, BOUNDARY)                   \
    template<>                                                                  \
    System<CUDASimulatorTraits<REAL, BOUNDARY>>::System(                        \
            const std::size_t num_particles, const boundary_type& bd)           \
        : velocity_initialized_(false), boundary_(bd), topology_(num_particles),\
          attributes_(), num_particles_(num_particles),                         \
          masses_     (num_particles), rmasses_     (num_particles),            \
          positions_  (num_particles), velocities_  (num_particles),            \
          forces_     (num_particles),                                          \
          masses_d_   (num_particles), rmasses_d_   (num_particles),            \
          positions_d_(num_particles), velocities_d_(num_particles),            \
          forces_d_   (num_particles)                                           \
    {} /**/

GENERATE_SYSTEM_CONSTRUCTOR_INSTANCES(double, UnlimitedBoundary)
GENERATE_SYSTEM_CONSTRUCTOR_INSTANCES(float,  UnlimitedBoundary)
GENERATE_SYSTEM_CONSTRUCTOR_INSTANCES(double, CuboidalPeriodicBoundary)
GENERATE_SYSTEM_CONSTRUCTOR_INSTANCES(float,  CuboidalPeriodicBoundary)

#undef GENERATE_SYSTEM_CONSTRUCTOR_INSTANCES

template<> System<CUDASimulatorTraits<double, UnlimitedBoundary>>::~System(){}
template<> System<CUDASimulatorTraits<float,  UnlimitedBoundary>>::~System(){}
template<> System<CUDASimulatorTraits<double, CuboidalPeriodicBoundary>>::~System(){}
template<> System<CUDASimulatorTraits<float,  CuboidalPeriodicBoundary>>::~System(){}

#define GENERATE_SYSTEM_COPY_CONSTRUCTOR_INSTANCES(REAL, BOUNDARY)              \
    template<>                                                                  \
    System<CUDASimulatorTraits<REAL, BOUNDARY>>::System(const System& other)    \
        : velocity_initialized_(other.velocity_initialized_),                   \
          boundary_(other.boundary_), topology_(other.topology_),               \
          attributes_(other.attributes_), num_particles_(other.num_particles_), \
          masses_     (other.masses_     ), rmasses_     (other.rmasses_     ), \
          positions_  (other.positions_  ), velocities_  (other.velocities_  ), \
          forces_     (other.forces_     ),                                     \
          masses_d_   (other.masses_d_   ), rmasses_d_   (other.rmasses_d_   ), \
          positions_d_(other.positions_d_), velocities_d_(other.velocities_d_), \
          forces_d_   (other.forces_d_   )                                      \
    {} /**/

GENERATE_SYSTEM_COPY_CONSTRUCTOR_INSTANCES(double, UnlimitedBoundary)
GENERATE_SYSTEM_COPY_CONSTRUCTOR_INSTANCES(float,  UnlimitedBoundary)
GENERATE_SYSTEM_COPY_CONSTRUCTOR_INSTANCES(double, CuboidalPeriodicBoundary)
GENERATE_SYSTEM_COPY_CONSTRUCTOR_INSTANCES(float,  CuboidalPeriodicBoundary)

#undef GENERATE_SYSTEM_COPY_CONSTRUCTOR_INSTANCES

#define GENERATE_SYSTEM_MOVE_CONSTRUCTOR_INSTANCES(REAL, BOUNDARY)                                     \
    template<>                                                                                         \
    System<CUDASimulatorTraits<REAL, BOUNDARY>>::System(System&& other)                                \
        : velocity_initialized_(other.velocity_initialized_),                                          \
          boundary_   (std::move(other.boundary_)),    topology_     (std::move(other.topology_)),     \
          attributes_ (std::move(other.attributes_ )), num_particles_(std::move(other.num_particles_)),\
          masses_     (std::move(other.masses_     )), rmasses_      (std::move(other.rmasses_      )),\
          positions_  (std::move(other.positions_  )), velocities_   (std::move(other.velocities_   )),\
          forces_     (std::move(other.forces_     )),                                                 \
          masses_d_   (std::move(other.masses_d_   )), rmasses_d_    (std::move(other.rmasses_d_    )),\
          positions_d_(std::move(other.positions_d_)), velocities_d_ (std::move(other.velocities_d_ )),\
          forces_d_   (std::move(other.forces_d_   ))                                                  \
    {} /**/

GENERATE_SYSTEM_MOVE_CONSTRUCTOR_INSTANCES(double, UnlimitedBoundary)
GENERATE_SYSTEM_MOVE_CONSTRUCTOR_INSTANCES(float,  UnlimitedBoundary)
GENERATE_SYSTEM_MOVE_CONSTRUCTOR_INSTANCES(double, CuboidalPeriodicBoundary)
GENERATE_SYSTEM_MOVE_CONSTRUCTOR_INSTANCES(float,  CuboidalPeriodicBoundary)

#undef GENERATE_SYSTEM_MOVE_CONSTRUCTOR_INSTANCES

#define GENERATE_SYSTEM_COPY_ASSIGNER_INSTANCES(REAL, BOUNDARY)                 \
    template<>                                                                  \
    System<CUDASimulatorTraits<REAL, BOUNDARY>>&                                \
    System<CUDASimulatorTraits<REAL, BOUNDARY>>::operator=(const System& other) \
    {                                                                           \
        this->velocity_initialized_ = other.velocity_initialized_;              \
        this->boundary_      = other.boundary_     ;                            \
        this->topology_      = other.topology_     ;                            \
        this->attributes_    = other.attributes_   ;                            \
        this->num_particles_ = other.num_particles_;                            \
        this->masses_        = other.masses_       ;                            \
        this->rmasses_       = other.rmasses_      ;                            \
        this->positions_     = other.positions_    ;                            \
        this->velocities_    = other.velocities_   ;                            \
        this->forces_        = other.forces_       ;                            \
        this->masses_d_      = other.masses_d_     ;                            \
        this->rmasses_d_     = other.rmasses_d_    ;                            \
        this->positions_d_   = other.positions_d_  ;                            \
        this->velocities_d_  = other.velocities_d_ ;                            \
        this->forces_d_      = other.forces_d_     ;                            \
        return *this;                                                           \
    } /**/

GENERATE_SYSTEM_COPY_ASSIGNER_INSTANCES(double, UnlimitedBoundary)
GENERATE_SYSTEM_COPY_ASSIGNER_INSTANCES(float,  UnlimitedBoundary)
GENERATE_SYSTEM_COPY_ASSIGNER_INSTANCES(double, CuboidalPeriodicBoundary)
GENERATE_SYSTEM_COPY_ASSIGNER_INSTANCES(float,  CuboidalPeriodicBoundary)

#undef GENERATE_SYSTEM_COPY_ASSIGNER_INSTANCES

#define GENERATE_SYSTEM_MOVE_ASSIGNER_INSTANCES(REAL, BOUNDARY)            \
    template<>                                                             \
    System<CUDASimulatorTraits<REAL, BOUNDARY>>&                           \
    System<CUDASimulatorTraits<REAL, BOUNDARY>>::operator=(System&& other) \
    {                                                                      \
        this->velocity_initialized_ = other.velocity_initialized_;         \
        this->boundary_      = std::move(other.boundary_     );            \
        this->topology_      = std::move(other.topology_     );            \
        this->attributes_    = std::move(other.attributes_   );            \
        this->num_particles_ = std::move(other.num_particles_);            \
        this->masses_        = std::move(other.masses_       );            \
        this->rmasses_       = std::move(other.rmasses_      );            \
        this->positions_     = std::move(other.positions_    );            \
        this->velocities_    = std::move(other.velocities_   );            \
        this->forces_        = std::move(other.forces_       );            \
        this->masses_d_      = std::move(other.masses_d_     );            \
        this->rmasses_d_     = std::move(other.rmasses_d_    );            \
        this->positions_d_   = std::move(other.positions_d_  );            \
        this->velocities_d_  = std::move(other.velocities_d_ );            \
        this->forces_d_      = std::move(other.forces_d_     );            \
        return *this;                                                      \
    } /**/

GENERATE_SYSTEM_MOVE_ASSIGNER_INSTANCES(double, UnlimitedBoundary)
GENERATE_SYSTEM_MOVE_ASSIGNER_INSTANCES(float,  UnlimitedBoundary)
GENERATE_SYSTEM_MOVE_ASSIGNER_INSTANCES(double, CuboidalPeriodicBoundary)
GENERATE_SYSTEM_MOVE_ASSIGNER_INSTANCES(float,  CuboidalPeriodicBoundary)

#undef GENERATE_SYSTEM_MOVE_ASSIGNER_INSTANCES

// ---------------------------------------------------------------------------
// class template instanciation

template class System<CUDASimulatorTraits<double, UnlimitedBoundary>>;
template class System<CUDASimulatorTraits<float,  UnlimitedBoundary>>;
template class System<CUDASimulatorTraits<double, CuboidalPeriodicBoundary>>;
template class System<CUDASimulatorTraits<float,  CuboidalPeriodicBoundary>>;

} // mjolnir
